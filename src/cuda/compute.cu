#include "hip/hip_runtime.h"
#include "compute.h"

#include <thrust/complex.h>

#include <cstdint>

#define MAX_ITER 100

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void kernel(
		double re_start, double re_step,
		double im_start, double im_step,
		int x, int y,
		uint8_t* buffer
) {

	printf("heloo!!!??");

	int i, j;
	i = blockIdx.x;
	j = threadIdx.x;

	double point_re = re_start + re_step * x;
	double point_im = im_start + im_step * y;

	thrust::complex<double> c(point_re, point_im);
	thrust::complex<double> z(0, 0);

	auto iter = 0;
	while (thrust::abs(z) <= 2 && iter < MAX_ITER) {
		z = z * z + c;
		iter++;
	}

	int addr = 3 * (j * x + i);
	buffer[addr] = 255;
	buffer[addr + 1] = 255;
	buffer[addr + 1] = 255;
}


void cuda_compute(
		double re_start, double re_end,
		double im_start, double im_end,
		int x, int y,
		uint8_t* buffer
	) {

	int bufferSize = x * y * 3 * sizeof(uint8_t);
	uint8_t *cudaBuffer;
	gpuErrchk(hipMalloc(&cudaBuffer, bufferSize));

	const double re_step = (re_end - re_start) / x;
	const double im_step = (im_end - im_start) / y;

  kernel<<<100, 100>>>(re_start, re_step, im_start, im_step, x, y, cudaBuffer);

	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

  hipMemcpy(buffer, cudaBuffer, bufferSize, hipMemcpyDeviceToHost);

  hipFree(cudaBuffer);
}
