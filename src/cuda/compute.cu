#include "hip/hip_runtime.h"
#include "compute.h"

#include <thrust/complex.h>

#include <cstdint>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void kernel(
		double re_start, double re_step,
		double im_start, double im_step,
		int start_x, int start_y,
		int x, int y,
		uint8_t* buffer,
    int max_iter
) {
	const auto i = start_x + blockIdx.x;
	const auto j = start_y + threadIdx.x;

	if (i >= x || j >= y)
		return;

	const double point_re = re_start + re_step * i;
	const double point_im = im_start + im_step * j;

	const thrust::complex<double> c(point_re, point_im);
	thrust::complex<double> z(0, 0);

	auto iter = 0;
	while (thrust::abs(z) <= 200 && iter < max_iter) {
		z = z * z + c;
		iter++;
	}

	float gradient = (float)(iter * iter) / (max_iter * max_iter);
	int addr = 3 * (j * x + i);

	buffer[addr]     =  70 * gradient;
	buffer[addr + 1] =  20 * gradient;
	buffer[addr + 2] = 250 * gradient;
}

const int kern = 720;

void cuda_compute(
		double re_start, double re_end,
		double im_start, double im_end,
		int x, int y,
		uint8_t* buffer,
    int max_iter
	) {

	int bufferSize = x * y * 3 * sizeof(uint8_t);
	uint8_t *cudaBuffer;
	hipMalloc(&cudaBuffer, bufferSize);

	const double re_step = (re_end - re_start) / x;
	const double im_step = (im_end - im_start) / y;

	auto i = 0;
	while (i < x) {
		auto j = 0;
		while (j < y) {
			kernel<<<kern, kern>>>(re_start, re_step, im_start, im_step, i, j, x, y, cudaBuffer, max_iter);
			j += kern;
		}
		i += kern;
	}

	hipDeviceSynchronize();

  hipMemcpy(buffer, cudaBuffer, bufferSize, hipMemcpyDeviceToHost);

  hipFree(cudaBuffer);
}
